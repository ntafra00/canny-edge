#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <algorithm>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

using namespace std;

const char *inputImagePath = "../input.jpg";
const char *outputImagePath = "output-gpu.jpg";

int *imgToArray(uint8_t *pixelPtr, int sizeRows, int sizeCols, int sizeDepth)
{
    int *pixels = (int *)malloc((sizeRows * sizeCols * sizeDepth) * sizeof(int));
    for (int i = 0; i < sizeRows; i++)
    {
        for (int j = 0; j < sizeCols; j++)
        {
            for (int k = 0; k < sizeDepth; k++)
            {
                // converting BGR to RGB colors
                pixels[i * sizeCols * sizeDepth + j * sizeDepth + k] =
                    (int)pixelPtr[i * sizeCols * sizeDepth + j * sizeDepth + 2 - k];
            }
        }
    }
    return pixels;
}

void arrayToImg(int *pixels, uint8_t *pixelPtr, int sizeRows, int sizeCols, int sizeDepth)
{
    for (int i = 0; i < sizeRows; i++)
    {
        for (int j = 0; j < sizeCols; j++)
        {
            for (int k = 0; k < sizeDepth; k++)
            {
                pixelPtr[i * sizeCols * sizeDepth + j * sizeDepth + k] =
                    (uint8_t)pixels[i * sizeCols * sizeDepth + j * sizeDepth + (sizeDepth - 1 - k)];
            }
        }
    }
    return;
}

std::vector<int> gaussianBlur(std::vector<int> &pixels, std::vector<std::vector<double>> &kernel, double kernelConst, int sizeRows, int sizeCols, int sizeDepth)
{
    std::vector<int> pixelsBlur(sizeRows * sizeCols * sizeDepth);
    for (int i = 0; i < sizeRows; i++)
    {
        for (int j = 0; j < sizeCols; j++)
        {
            for (int k = 0; k < sizeDepth; k++)
            {
                double sum = 0;
                double sumKernel = 0;
                for (int y = -2; y <= 2; y++)
                {
                    for (int x = -2; x <= 2; x++)
                    {
                        if ((i + x) >= 0 && (i + x) < sizeRows && (j + y) >= 0 && (j + y) < sizeCols)
                        {
                            double channel = (double)pixels[(i + x) * sizeCols * sizeDepth + (j + y) * sizeDepth + k];
                            sum += channel * kernelConst * kernel[x + 2][y + 2];
                            sumKernel += kernelConst * kernel[x + 2][y + 2];
                        }
                    }
                }
                pixelsBlur[i * sizeCols * sizeDepth + j * sizeDepth + k] = (int)(sum / sumKernel);
            }
        }
    }
    return pixelsBlur;
}

std::vector<int> rgbToGrayscale(std::vector<int> &pixels, int sizeRows, int sizeCols, int sizeDepth)
{
    std::vector<int> pixelsGray(sizeRows * sizeCols);
    for (int i = 0; i < sizeRows; i++)
    {
        for (int j = 0; j < sizeCols; j++)
        {
            int sum = 0;
            for (int k = 0; k < sizeDepth; k++)
            {
                sum = sum + pixels[i * sizeCols * sizeDepth + j * sizeDepth + k];
            }
            pixelsGray[i * sizeCols + j] = (int)(sum / sizeDepth);
        }
    }
    return pixelsGray;
}

__global__ void gaussianBlur(int *inputPixels, int *blurredPixels, int sizeRows, int sizeCols, int sizeDepth)
{
    double kernel[5][5] = {{2.0, 4.0, 5.0, 4.0, 2.0},
                           {4.0, 9.0, 12.0, 9.0, 4.0},
                           {5.0, 12.0, 15.0, 12.0, 5.0},
                           {4.0, 9.0, 12.0, 9.0, 4.0},
                           {2.0, 4.0, 5.0, 4.0, 2.0}};
    double kernelConst = (1.0 / 159.0);

    int i = blockIdx.x * blockDim.x + threadIdx.x; // x-coordinate of the pixel
    int j = blockIdx.y * blockDim.y + threadIdx.y; // y-coordinate of the pixel
    int k = blockIdx.z * blockDim.z + threadIdx.z; // z-coordinate of the pixel

    if (i < sizeRows && j < sizeCols && k < sizeDepth)
    {
        double sum = 0;
        double sumKernel = 0;
        for (int y = -2; y <= 2; y++)
        {
            for (int x = -2; x <= 2; x++)
            {
                if ((i + x) >= 0 && (i + x) < sizeRows && (j + y) >= 0 && (j + y) < sizeCols)
                {
                    double channel = (double)inputPixels[(i + x) * sizeCols * sizeDepth + (j + y) * sizeDepth + k];
                    sum += channel * kernelConst * kernel[x + 2][y + 2];
                    sumKernel += kernelConst * kernel[x + 2][y + 2];
                }
            }
        }
        blurredPixels[i * sizeCols * sizeDepth + j * sizeDepth + k] = (int)(sum / sumKernel);
    }
}

__global__ void rgbToGrayscale(int *blurredPixels, int *grayscaledPixels, int sizeRows, int sizeCols)
{
}

void cannyEdgeDetection(uint8_t *inputImage, double lowerThreshold, double higherThreshold, int width, int height, int channels)
{
    int *pixels = imgToArray(inputImage, height, width, channels);

    // hipMalloc((void **)&pixelsPtr, height * width * channels * sizeof(int));

    // hipMemcpy(pixelsPtr, pixels, height * width * channels * sizeof(int), hipMemcpyHostToDevice);

    // dim3 blockSize(16, 16);
    // dim3 numBlocks((width + blockSize.x - 1) / blockSize.x,
    //                (height + blockSize.y - 1) / blockSize.y,
    //                (channels + blockSize.z - 1) / blockSize.z);

    // // GAUSSIAN_BLUR:

    // gaussianBlur<<<numBlocks, blockSize>>>(pixelsPtr, pixelsPtr, height, width, channels);

    // // rgbToGrayscale<<<numBlocks, threadsPerBlock>>>(pixelsPtr, pixelsPtr, height, width);

    // hipMemcpy(pixels, pixelsPtr, height * width * channels * sizeof(int), hipMemcpyDeviceToHost);

    // std::vector<int>
    //     pixelsBlur = gaussianBlur(pixels, kernel, kernelConst, height, width, channels);

    // // GRAYSCALE:

    // std::vector<int> pixelsGray = rgbToGrayscale(pixelsBlur, height, width, channels);

    // // CANNY_FILTER:

    // std::vector<int> pixelsCanny = cannyFilter(pixelsGray, height, width, 1, lowerThreshold, higherThreshold);

    uint8_t *outputImage = (unsigned char *)malloc(width * height);
    arrayToImg(pixelsCanny, outputImage, height, width, 1);

    stbi_write_jpg(outputImagePath, width, height, 1, outputImage, 100);

    free(outputImage);
}

int main()
{
    int width, height, channels;

    // Load the input image
    uint8_t *inputImage = stbi_load(inputImagePath, &width, &height, &channels, STBI_rgb);
    if (inputImage == NULL)
    {
        printf("Failed to load image: %s\n", inputImagePath);
        return -1;
    }

    // if (width < 1400 || height < 1400)
    // {
    //     printf("Choose image with greater resolution\n");
    //     return -1;
    // }

    if (channels != 3)
    {
        printf("Images is not in RGB format\n");
        return -1;
    }

    double lowerThreshold = 0.03;
    double higherThreshold = 0.1;

    cannyEdgeDetection(inputImage, lowerThreshold, higherThreshold, width, height, channels);

    return 0;
}